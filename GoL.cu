
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <unistd.h>
#include <string>

using namespace std;

void printBoard(char array[], int size, int width);
__global__
void getNeighbours(char startBoard[], char finalBoard[], int width, int height);

int main(int argc, char** argv){

	const char* filename;
	if(argc == 4 || argc == 5){
		string arg1 = argv[1];
		string arg3 = argv[3];
		if(argc == 4 && arg1 == "-i"){
			filename = argv[3];
		}
		else if(argc == 5 && arg1 == "-i" && arg3 == "-v"){
			filename = argv[4];
		}
		else{
			cout << "Wrong arguments given" << endl;
			return 0;
		}
	}
	else{
		cout << "Wrong number of arguments given" << endl;
		return 0;
	}


	int numIter = atoi(argv[2]);

	int width = 0;
	int height = 0;

	ifstream infile;
	infile.open(filename);

	//Getting width and Height from file
	string line;
	while(getline(infile, line)){
		if(width == 0){
			width = line.length();
		}
		if(line.length() == width){
			height ++;
		}
	}
	int arraySize = height * width;
	infile.clear();
	infile.seekg(0, infile.beg);

	//Creating arrays
	char* startBoard;
	char* finalBoard;
	char world[arraySize];

	for (int i = 0; i < arraySize; ++i)
	{
		infile >> world[i];
	}

	//Printing Start Board
	printBoard(world, arraySize, width);
	cout << endl;


	//Device arrays
	hipMalloc((void**)&startBoard, height * width * sizeof(char));
	hipMalloc((void**)&finalBoard, height * width * sizeof(char));

	hipMemcpy(startBoard, world, height * width * sizeof(char), hipMemcpyHostToDevice);


	//Number of iterations
	for(int iter = 0; iter < numIter; iter++){

		int blockSize = 1024;
		int numBlocks = (arraySize + blockSize -1) / blockSize;

		getNeighbours<<<numBlocks, blockSize>>>(startBoard, finalBoard, width, height);

		swap(startBoard,finalBoard);

		//Printing each iteration board

		if(argc == 5){
			hipMemcpy(world, startBoard, height * width * sizeof(char), hipMemcpyDeviceToHost);
			printBoard(world, arraySize, width);
			cout << endl;
		}

		unsigned int microseconds;
		microseconds = 100000;
		//usleep(microseconds);


	}

	//Print only final iteration
	if(argc == 4){
		hipMemcpy(world, startBoard, height * width * sizeof(char), hipMemcpyDeviceToHost);
		printBoard(world, arraySize, width);
	}



	hipFree(startBoard);
	hipFree(finalBoard);
	return 0;
}

void printBoard(char array[], int size, int width){
	int count = 0;
	for (int i = 0; i < size; ++i)
	{
		cout << array[i];
		count ++;
		if(count == width){
			cout << endl;
			count = 0;
		}
	}
}

__global__
void getNeighbours(char startBoard[], char finalBoard[], int width, int height){

	//Finding Neighbours
	int index;
	int x;
	int y;

	int currentCell = blockIdx.x * blockDim.x + threadIdx.x;

	if(currentCell < width * height){
		y = currentCell / width;
		x = currentCell - (width * y);
		int neighbours = 0;
			//Checking surrounding squares
			for (int i = y - 1; i <= y + 1; i++)
			{
				for (int j = x - 1; j <= x + 1; j++){
					if ( j == x && i == y ) {
						continue;
					}
					//Check if on board
					else if(j > -1 && j < width && i > -1 && i < height){
						index = width * i + j;
						if(startBoard[index] == 'X'){
							neighbours ++;
						}
					}
					//Handle wrap around and add neighbours
					else{
						int jTemp = j;
						int iTemp = i;

						if(j == -1){
							jTemp = width - 1;
						}
						if(j == width){
							jTemp = 0;
						}
						if(i == -1){
							iTemp = height - 1;
						}
						if(i == height){
							iTemp = 0;
						}
						index = width * iTemp + jTemp;
						if(startBoard[index] == 'X'){
							neighbours ++;
						}
					}

				}
			}
			if(neighbours == 3 || startBoard[currentCell] == 'X' && neighbours == 2){
				finalBoard[currentCell] = 'X';
			}
			else{
				finalBoard[currentCell] = '-';
			}
	}
}


